#include "hip/hip_runtime.h"
#include <iostream>

#define BS 17

using namespace std;

__global__ void bfs(int* dflag, int* dlevel, int* dvertex_pointers, int* dedges, int n){
    int tid=blockDim.x*blockIdx.x+threadIdx.x;
    int e;
    
    if(tid<n){
        for(int i=dvertex_pointers[tid];i<dvertex_pointers[tid+1];i++){
            e=dedges[i];
            if(dlevel[e]>(dlevel[tid]+1)){
                dlevel[e]=dlevel[tid]+1;
                *dflag=1;
            }
        }
    }
}

__global__ void truncated_bfs(int* dlevel, int* dvertex_pointers, int* dedges, int* ddist, int n, int u, int* dflag, int* dreached){
    int tid=blockDim.x*blockIdx.x+threadIdx.x;
    int e;
    
    if(tid<n && tid!=u){
        for(int i=dvertex_pointers[tid];i<dvertex_pointers[tid+1];i++){
            e=dedges[i];
            if(e==u){
                continue;
            }
            
            if(ddist[e]>(ddist[tid]+1)){
                if(dlevel[e]<=dlevel[u]){
                    *dreached=1;
                }
                ddist[e]=ddist[tid]+1;
                *dflag=1;
            }
        }
    }
}

__global__ void find_cut_vertices(int* dlevel, int* dvertex_pointers, int* dedges, int* dcut_vertex, int n, int root, int* dunsafe_vertex){
    int tid=blockDim.x*blockIdx.x+threadIdx.x;
    int threadsPerBlock=BS;
    int blocksPerGrid=(n+BS-1)/BS;

    if(tid<n && tid!=root){
        for(int j=dvertex_pointers[tid];j<dvertex_pointers[tid+1];j++){
            
            int e=dedges[j];

            int* dflag;
            int* ddist;
            hipMalloc((void**) &dflag,sizeof(int));
            int* dreached;
            hipMalloc((void**) &dreached,sizeof(int));
            hipMalloc((void**) &ddist,sizeof(int)*n);
            for(int i=0;i<n;i++){
                ddist[i]=INT_MAX/2;
            }
            ddist[e]=0;
            *dflag=1;
            *dreached=0;
            while(*dflag && !(*dreached)){
                *dflag=0;
                truncated_bfs<<<1,n>>>(dlevel,dvertex_pointers,dedges,ddist,n,tid,dflag,dreached);
                if (hipSuccess != hipDeviceSynchronize()) {
                    return;
                }
            }
            if (hipSuccess != hipDeviceSynchronize()) {
                return;
            }
            if(!(*dreached)){
                dcut_vertex[tid]=1;
                dunsafe_vertex[e]=tid;
            }
            if(tid==root){
                break;
            }
        }
    }
    else if(tid==root){
        int e=dedges[dvertex_pointers[tid]];
        int* dflag;
        int* ddist;
        hipMalloc((void**) &dflag,sizeof(int));
        hipMalloc((void**) &ddist,sizeof(int)*n);
        int* dreached;
        hipMalloc((void**) &dreached,sizeof(int));
        bool done=false;
        for(int i=0;i<n;i++){
            ddist[i]=INT_MAX/2;
        }
        ddist[e]=0;
        *dflag=1;
        *dreached=0;
        while(*dflag){
            *dflag=0;
            truncated_bfs<<<1,n>>>(ddist,dvertex_pointers,dedges,ddist,n,tid,dflag,dreached);
            if (hipSuccess != hipDeviceSynchronize()) {
                return;
            }
        }
        for(int i=0;i<n;i++){
            if(ddist[i]>=INT_MAX/2 && i!=tid){
                dcut_vertex[tid]=1;
                dunsafe_vertex[e]=tid;
                break;
            }
        }
    }
}

__global__ void find_minimum(int* dlevel, int* dvertex_pointers, int* dedges, int* ddist, int n, int* dflag, int* dcurrent_cut_vertex, int* dminimum, int cur_level, int* dvisited){
    int tid=blockDim.x*blockIdx.x+threadIdx.x;
    int threadsPerBlock=BS;
    int blocksPerGrid=(n+BS-1)/BS;

    int e;
    if(dlevel[tid]>=(cur_level-1) && tid!=*dcurrent_cut_vertex){
        for(int i=dvertex_pointers[tid];i<dvertex_pointers[tid+1];i++){
            if(dvisited[i]){
                continue;
            }
            e=dedges[i];

            if(tid!=*dcurrent_cut_vertex){
                atomicMin(dminimum,tid);
            }
            
            if(ddist[e]>(ddist[tid]+1)){
                if(dlevel[e]<dlevel[*dcurrent_cut_vertex]){
                    continue;
                }
                ddist[e]=ddist[tid]+1;
                if(e!=*dcurrent_cut_vertex){
                    atomicMin(dminimum,e);
                }
                *dflag=1;
            }
        }
    }
}

__global__ void set_bcc_id(int* dlevel, int* dvertex_pointers, int* dedges, int* ddist, int n, int* dflag, int cur_level, int* dminimum, int* dcurrent_cut_vertex, int* dvisited, int* dbcc){
    int tid=blockDim.x*blockIdx.x+threadIdx.x;
    int threadsPerBlock=BS;
    int blocksPerGrid=(n+BS-1)/BS;

    int e;
    if(dlevel[tid]>=(cur_level-1)){
        for(int i=dvertex_pointers[tid];i<dvertex_pointers[tid+1];i++){
            if(dvisited[i]){
                continue;
            }
            dvisited[i]=1;
            e=dedges[i];
            
            if(ddist[e]>(ddist[tid]+1)){
                if(dlevel[e]<dlevel[*dcurrent_cut_vertex]){
                    continue;
                }
                ddist[e]=ddist[tid]+1;
                dbcc[i]=*dminimum;
                // printf("%d %d\n",*dminimum,i);
                *dflag=1;
            }
        }
    }
}

__global__ void find_bcc(int* dlevel, int* dvertex_pointers, int* dedges, int* dunsafe_vertex, int cur_level, int n, int* dbcc, int* dvisited){
    int tid=blockDim.x*blockIdx.x+threadIdx.x;
    int threadsPerBlock=BS;
    int blocksPerGrid=(n+BS-1)/BS;

    if(dlevel[tid]==cur_level && dunsafe_vertex[tid]!=-1){
        int* dminimum;
        hipMalloc(&dminimum,sizeof(int));
        *dminimum=INT_MAX/2;

        int* dcurrent_cut_vertex;
        hipMalloc(&dcurrent_cut_vertex,sizeof(int));
        *dcurrent_cut_vertex=dunsafe_vertex[tid];
        int* dflag;
        int* ddist;
        hipMalloc((void**) &dflag,sizeof(int));
        hipMalloc((void**) &ddist,sizeof(int)*n);
        for(int i=0;i<n;i++){
            ddist[i]=INT_MAX/2;
        }
        ddist[tid]=0;
        *dflag=1;
        while(*dflag){
            *dflag=0;
            find_minimum<<<1,n>>>(dlevel,dvertex_pointers,dedges,ddist,n,dflag,dcurrent_cut_vertex,dminimum,cur_level,dvisited);
            if (hipSuccess != hipDeviceSynchronize()) {
                return;
            }
        }
        printf("%d\n",*dminimum);

        for(int i=0;i<n;i++){
            ddist[i]=INT_MAX/2;
        }
        ddist[tid]=0;
        *dflag=1;
        while(*dflag){
            *dflag=0;
            set_bcc_id<<<1,n>>>(dlevel,dvertex_pointers,dedges,ddist,n,dflag,cur_level,dminimum,dcurrent_cut_vertex,dvisited,dbcc);
            if (hipSuccess != hipDeviceSynchronize()) {
                return;
            }
        }
    }
}

int main(){
    int n=17;
    int m=50;
    int threadsPerBlock=BS;
    int blocksPerGrid=(n+BS-1)/BS;

    int vertex_pointers[18];
    int edges[50]={1,2,0,2,5,0,1,3,5,4,2,12,11,2,5,1,2,4,6,7,7,5,5,6,8,10,7,10,10,7,8,9,12,3,13,3,11,13,14,11,12,14,15,13,12,16,14,16,14,15};
    vertex_pointers[0]=0;
    vertex_pointers[1]=2;
    vertex_pointers[2]=5;
    vertex_pointers[3]=10;
    vertex_pointers[4]=13;
    vertex_pointers[5]=15;
    vertex_pointers[6]=20;
    vertex_pointers[7]=22;
    vertex_pointers[8]=26;
    vertex_pointers[9]=28;
    vertex_pointers[10]=29;
    vertex_pointers[11]=32;
    vertex_pointers[12]=35;
    vertex_pointers[13]=39;
    vertex_pointers[14]=42;
    vertex_pointers[15]=46;
    vertex_pointers[16]=48;
    vertex_pointers[17]=50;

    int root=0;

    int* dvertex_pointers;
    hipMalloc((void**)&dvertex_pointers,sizeof(int)*(n+1));
    hipMemcpy(dvertex_pointers,vertex_pointers,sizeof(int)*(n+1),hipMemcpyHostToDevice);

    int* dedges;
    hipMalloc((void**)&dedges,sizeof(int)*m);
    hipMemcpy(dedges,edges,sizeof(int)*m,hipMemcpyHostToDevice);

    int cut_vertex[17]={0};
    int* dcut_vertex;
    hipMalloc((void**) &dcut_vertex,sizeof(int)*n);
    hipMemcpy(dcut_vertex,cut_vertex,sizeof(int)*n,hipMemcpyHostToDevice);

    int level[17];
    int unsafe_vertex[17];
    for(int i=0;i<n;i++){
        level[i]=INT_MAX/2;
        unsafe_vertex[i]=-1;
    }
    level[root]=0;
    int* dlevel;
    hipMalloc((void**) &dlevel,sizeof(int)*n);

    int* dunsafe_vertex;
    hipMalloc((void**) &dunsafe_vertex,sizeof(int)*n);

    int flag=1;
    int* dflag;
    hipMalloc((void**) &dflag,sizeof(int));

    int max_level=0;
    while(flag){
        flag=0;
        hipMemcpy(dlevel,level,sizeof(int)*n,hipMemcpyHostToDevice);
        hipMemcpy(dflag,&flag,sizeof(int),hipMemcpyHostToDevice);
        bfs<<<1,n>>>(dflag,dlevel,dvertex_pointers,dedges,n);
        max_level++;
        hipMemcpy(level,dlevel,sizeof(int)*n,hipMemcpyDeviceToHost);
        hipMemcpy(&flag,dflag,sizeof(int),hipMemcpyDeviceToHost);
    }
    hipDeviceSynchronize();

    hipMemcpy(dunsafe_vertex,unsafe_vertex,sizeof(int)*n,hipMemcpyHostToDevice);
    hipMemcpy(dlevel,level,sizeof(int)*n,hipMemcpyHostToDevice);
    find_cut_vertices<<<1,n>>>(dlevel,dvertex_pointers,dedges,dcut_vertex,n,root,dunsafe_vertex);
    hipDeviceSynchronize();
    hipMemcpy(&cut_vertex,dcut_vertex,sizeof(int)*n,hipMemcpyDeviceToHost);
    hipMemcpy(unsafe_vertex,dunsafe_vertex,sizeof(int)*n,hipMemcpyDeviceToHost);

    // for(int i=0;i<n;i++){
    //     if(unsafe_vertex[i]!=-1){
    //         cout<<i<<" "<<unsafe_vertex[i]<<endl;
    //     }
    // }

    int bcc[50];
    for(int i=0;i<n;i++){
        bcc[i]=-1;
    }
    int* dbcc;
    hipMalloc(&dbcc,sizeof(int)*m);

    int visited[50]={0};
    int* dvisited;
    hipMalloc(&dvisited,sizeof(int)*m);

    for(int i=max_level;i>=0;i--){
        hipMemcpy(dvisited,visited,sizeof(int)*m,hipMemcpyHostToDevice);
        hipMemcpy(dbcc,bcc,sizeof(int)*m,hipMemcpyHostToDevice);
        find_bcc<<<1,n>>>(dlevel,dvertex_pointers,dedges,dunsafe_vertex,i,n,dbcc,dvisited);
        hipDeviceSynchronize();
        hipMemcpy(bcc,dbcc,sizeof(int)*m,hipMemcpyDeviceToHost);
        hipMemcpy(visited,dvisited,sizeof(int)*m,hipMemcpyDeviceToHost);
    }

    // for(int i=0;i<m;i++){
    //     cout<<i<<" "<<bcc[i]<<endl;
    // }
}