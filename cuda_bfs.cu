/*
* Program to perform Parallel BFS on a graph given in CSR format. 
* Output is the levels of each node in the graph with root vertex at level 0.
* To execute, run the following command on a system with CUDA support enabled GPUs
* bash run.sh cuda_bfs.cu
*/



#include <hip/hip_runtime.h>
#include <iostream>
#include <climits>
#include <chrono>

#define BS 1024
#define N 11
#define M 20
#define ROOT 0

using namespace std;

__global__ void bfs(int* dflag, int* ddist, int* dvertex_pointers, int* dedges){
    int tid=blockDim.x*blockIdx.x+threadIdx.x;
    int e;

    int *global_now;
    hipMalloc(&global_now,sizeof(int));

    clock_t start = clock();
    clock_t now;
    for (;;) {
    now = clock();
    clock_t cycles = now > start ? now - start : now + (0xffffffff - start);
        if (cycles >= 10000) {
            break;
        }
    }
    // Stored "now" in global memory here to prevent the
    // compiler from optimizing away the entire loop.
    *global_now = now;
    
    if(tid<N){
        for(int i=dvertex_pointers[tid];i<dvertex_pointers[tid+1];i++){
            e=dedges[i];
            if(ddist[e]>(ddist[tid]+1)){
                ddist[e]=ddist[tid]+1;
                *dflag=1;
            }
        }
    }
}

__global__ void initialize(int* ddist){
    int tid=blockDim.x*blockIdx.x+threadIdx.x;

    if(tid<N){
        ddist[tid]=INT_MAX/2;
    }
}

int main(){

    int threadsPerBlock=BS;
    int blocksPerGrid=(N+BS-1)/BS;

    int vertex_pointers[N+1];
    int edges[M]={7,1,9,10,0,3,2,6,7,5,7,8,4,3,3,0,4,4,0,0};
    vertex_pointers[0]=0;
    vertex_pointers[1]=4;
    vertex_pointers[2]=5;
    vertex_pointers[3]=6;
    vertex_pointers[4]=9;
    vertex_pointers[5]=12;
    vertex_pointers[6]=13;
    vertex_pointers[7]=14;
    vertex_pointers[8]=17;
    vertex_pointers[9]=18;
    vertex_pointers[10]=19;
    vertex_pointers[11]=20;

    // Start timer
    float time,total_time=0;
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    auto start_t = chrono::high_resolution_clock::now();

    int* dvertex_pointers;
    hipMalloc((void**)&dvertex_pointers,sizeof(int)*(N+1));
    hipMemcpy(dvertex_pointers,vertex_pointers,sizeof(int)*(N+1),hipMemcpyHostToDevice);

    int* dedges;
    hipMalloc((void**)&dedges,sizeof(int)*M);
    hipMemcpy(dedges,edges,sizeof(int)*M,hipMemcpyHostToDevice);

    int dist[N];

    int* ddist;
    hipMalloc((void**) &ddist,sizeof(int)*N);

    hipEventRecord(start,0);
    initialize<<<blocksPerGrid,threadsPerBlock>>>(ddist);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time,start,stop);
    total_time+=time;

    hipMemcpy(dist,ddist,sizeof(int)*N,hipMemcpyDeviceToHost);
    
    dist[ROOT]=0;

    int flag=1;
    int* dflag;
    hipMalloc(&dflag,sizeof(int));

    while(flag){
        flag=0;
        hipMemcpy(ddist,dist,sizeof(int)*N,hipMemcpyHostToDevice);
        hipMemcpy(dflag,&flag,sizeof(int),hipMemcpyHostToDevice);

        hipEventRecord(start,0);
        bfs<<<blocksPerGrid,threadsPerBlock>>>(dflag,ddist,dvertex_pointers,dedges);
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time,start,stop);

        hipMemcpy(dist,ddist,sizeof(int)*N,hipMemcpyDeviceToHost);
        hipMemcpy(&flag,dflag,sizeof(int),hipMemcpyDeviceToHost);
        total_time+=time;
    }
    
    auto stop_t = chrono::high_resolution_clock::now();

    hipEventDestroy(start);
    hipEventDestroy(stop);

    cout<<"Vertex Distance\n";
    for(int i=0;i<N;i++){
        printf("%4d  %7d\n",i+1,dist[i]);
    }

    auto duration = chrono::duration_cast<chrono::microseconds>(stop_t - start_t);
    printf("CPU Time Taken: %f ms\n", ((float) duration.count())/1000.0);

    printf("GPU Time Taken: %f ms\n", total_time);
    
}